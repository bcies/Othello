
#include <hip/hip_runtime.h>
extern "C" __global__ void playout(int *rands, int *numRands, int *board,
		int *boardWidth, int *colorToPlay, float *wins) {

	int l_width = *boardWidth;

	//NOTE!!!!! hardcoded!!! change if you can.....
	int tempBoard[8 * 8];
	for (int j = 0; j < l_width * l_width; j++) {
		tempBoard[j] = board[j];
	}

	int colorTP = *colorToPlay;
	int end = false;
	int black = 0;
	int white = 0;
	int count = 0;

	while (!end) {
		int n = rands[(blockIdx.x * (*numRands / blockDim.x) + count
				+ threadIdx.x) % *numRands];
		count += 1;

		//Check if n is a legal move
		//TODO


		if (tempBoard[n] == 0) {
			//play
			tempBoard[n] = colorTP;

			//capture
			int x = n % l_width;
			int y = n / l_width;
			int newX;
			int newY;
			int nextSpace;
			int captureFlag = false;
			int opposite;
			int i;
			if (colorTP == 1) {
				opposite = 2;
			} else {
				opposite = 1;
			}
			int numCapture = 0;

			//up
			if ((y < 6) && (board[x + (y + 1) * l_width] == opposite)) {
				newY = y + 2;
				numCapture += 1;
				while (newY <= 7) {
					nextSpace = board[x + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newY += 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + (y + i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// UpR
			if ((x < 6) && (y < 6)
					&& (board[x + 1 + (y + 1) * l_width] == opposite)) {
				numCapture += 1;
				newX = x + 2;
				newY = y + 2;
				while ((newX <= 7) && (newY <= 7)) {
					nextSpace = board[newX + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX += 1;
					newY += 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + i + (y + i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// R
			if ((x < 6) && (board[x + 1 + y * l_width] == opposite)) {
				numCapture += 1;
				newX = x + 2;
				while (newX <= 7) {
					nextSpace = board[newX + y * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX += 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + 1 + y * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// DoR
			if ((x < 6) && (y > 1)
					&& (board[x + 1 + (y - 1) * l_width] == opposite)) {
				numCapture += 1;
				newX = x + 2;
				newY = y - 2;
				while ((newX <= 7) && (newY >= 0)) {
					nextSpace = board[newX + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX += 1;
					newY -= 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + i + (y - i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// Do
			if ((y > 1) && (board[x + (y - 1) * l_width] == opposite)) {
				numCapture += 1;
				newY = y - 2;
				while (newY >= 0) {
					nextSpace = board[x + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newY -= 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + (y - i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// DoL
			if ((x > 1) && (y > 1)
					&& (board[x - 1 + (y - 1) * l_width] == opposite)) {
				numCapture += 1;
				newX = x - 2;
				newY = y - 2;
				while ((newX >= 0) && (newY >= 0)) {
					nextSpace = board[newX + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX -= 1;
					newY -= 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x - i + (y - i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// L
			if ((x > 1) && (board[x - 1 + y * l_width] == opposite)) {
				numCapture += 1;
				newX = x - 2;
				while (newX >= 0) {
					nextSpace = board[newX + y * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX -= 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x - i + y * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// UpL
			if ((x > 1) && (y < 6)
					&& (board[x - 1 + (y + 1) * l_width] == opposite)) {
				numCapture += 1;
				newX = x - 2;
				newY = y + 2;
				while ((newX >= 0) && (newY <= 7)) {
					nextSpace = board[newX + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX -= 1;
					newY += 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x - i + (y + i) * l_width] = colorTP;
			}

			//check end
			end = true;
			black = 0;
			white = 0;
			for (i = 0; i < l_width * l_width; i++) {
				if (tempBoard[i] == 0) {
					end = false;
					break;
				} else if (tempBoard[i] == 1) {
					black += 1;
				} else {
					white += 1;
				}
			}

		}
		if (colorTP == 1){
			colorTP =2;
		} else {
			colorTP = 1;
		}
	}
	black = 0;
	white = 0;
	for (int i = 0; i < l_width *l_width; i++){
		if (tempBoard[i] == 1){
			black += 1;
		} else if (tempBoard[i] == 2) {
			white += 1;
		}
	}
	int win = 0;
	if (black > white) {
		win = 1;
	} else if (white > black) {
		win = 2;
	}

	if (win == *colorToPlay) {
		atomicAdd(wins, (float) 1.0);
	} else if (win == 0) {
		atomicAdd(wins, (float) 0.5);
	}
}
extern "C"
__global__ void legalPlayout(int *rands, int *numRands, int *board,
		int *boardWidth, int *colorToPlay, float *wins) {

	int l_width = *boardWidth;

	//NOTE!!!!! hardcoded!!! change if you can.....
	int tempBoard[8 * 8];
	for (int j = 0; j < l_width * l_width; j++) {
		tempBoard[j] = board[j];
	}

	int colorTP = *colorToPlay;
	int end = false;
	int black = 0;
	int white = 0;
	int count = 0;

	int maxCount = *numRands / gridDim.x;

	while (count < (maxCount * 2) && !end) {
		int n = rands[(blockIdx.x * maxCount + count
				+ threadIdx.x) % *numRands];
		count += 1;

		//Check if n is a legal move
		//TODO
		int x = n % l_width;
		int y = n / l_width;
		int newX;
		int newY;
		int nextSpace;
		int captureFlag = false;
		int opposite;
		int i;
		if (colorTP == 1) {
			opposite = 2;
		} else {
			opposite = 1;
		}
		int legalMove = false;

		if (tempBoard[n] == 0) {
			// Up
			if ((y < 6) && (board[x + (y + 1) * l_width] == opposite)) {
				newY = y + 2;
				while (newY <= 7) {
					nextSpace = board[x + newY * l_width];
					if (nextSpace == colorTP) {
						legalMove = true;
						break;
					} else if (nextSpace == 0) {
						break;
					}
					newY += 1;
				}
			}
			//UpR
			if (!legalMove) {
				if ((x < 6) && (y < 6)
						&& (board[x + 1 + (y + 1) * l_width] == opposite)) {
					newX = x + 2;
					newY = y + 2;
					while ((newX <= 7) && (newY <= 7)) {
						nextSpace = board[newX + newY * l_width];
						if (nextSpace == colorTP) {
							legalMove = true;
							break;
						} else if (nextSpace == 0) {
							break;
						}
						newX += 1;
						;
						newY += 1;
					}
				}
			}
			//R
			if (!legalMove) {
				if ((x < 6) && (board[x + 1 + y * l_width] == opposite)) {
					newX = x + 2;
					while (newX <= 7) {
						nextSpace = board[newX + y * l_width];
						if (nextSpace == colorTP) {
							legalMove = true;
							break;
						} else if (nextSpace == 0) {
							break;
						}
						newX += 1;
					}
				}
			}
			//DoR
			if (!legalMove) {
				if ((x < 6) && (y > 1)
						&& (board[x + 1 + (y - 1) * l_width] == opposite)) {
					newX = x + 2;
					newY = y - 2;
					while ((newX <= 7) && (newY >= 0)) {
						nextSpace = board[newX + newY * l_width];
						if (nextSpace == colorTP) {
							legalMove = true;
							break;
						} else if (nextSpace == 0) {
							break;
						}
						newX += 1;
						newY -= 1;
					}
				}
			}
			//Do
			if (!legalMove) {
				if ((y > 1) && (board[x + (y - 1) * l_width] == opposite)) {
					newY = y - 2;
					while (newY >= 0) {
						nextSpace = board[x + newY * l_width];
						if (nextSpace == colorTP) {
							legalMove = true;
							break;
						} else if (nextSpace == 0) {
							break;
						}
						newY -= 1;
					}
				}
			}
			//DoL
			if (!legalMove) {
				if ((x > 1) && (y > 1)
						&& (board[x - 1 + (y - 1) * l_width] == opposite)) {
					newX = x - 2;
					newY = y - 2;
					while ((newX >= 0) && (newY >= 0)) {
						nextSpace = board[newX + newY * l_width];
						if (nextSpace == colorTP) {
							legalMove = true;
							break;
						} else if (nextSpace == 0) {
							break;
						}
						newX -= 1;
						newY -= 1;
					}
				}
			}
			//L
			if (!legalMove) {
				if ((x > 1) && (board[x - 1 + y * l_width] == opposite)) {
					newX = x - 2;
					while (newX >= 0) {
						nextSpace = board[newX + y * l_width];
						if (nextSpace == colorTP) {
							legalMove = true;
							break;
						} else if (nextSpace == 0) {
							break;
						}
						newX -= 1;
					}
				}
			}
			//UpL
			if (!legalMove) {
				if ((x > 1) && (y < 6)
						&& (board[x - 1 + (y + 1) * l_width] == opposite)) {
					newX = x - 2;
					newY = y + 2;
					while ((newX >= 0) && (newY <= 7)) {
						nextSpace = board[newX + newY * l_width];
						if (nextSpace == colorTP) {
							legalMove = true;
							break;
						} else if (nextSpace == 0) {
							break;
						}
						newX -= 1;
						newY += 1;
					}
				}
			}
		}

		if (legalMove) {
			//play
			tempBoard[n] = colorTP;

			//capture

			int numCapture = 0;

			//up
			if ((y < 6) && (board[x + (y + 1) * l_width] == opposite)) {
				newY = y + 2;
				numCapture += 1;
				while (newY <= 7) {
					nextSpace = board[x + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newY += 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + (y + i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// UpR
			if ((x < 6) && (y < 6)
					&& (board[x + 1 + (y + 1) * l_width] == opposite)) {
				numCapture += 1;
				newX = x + 2;
				newY = y + 2;
				while ((newX <= 7) && (newY <= 7)) {
					nextSpace = board[newX + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX += 1;
					newY += 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + i + (y + i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// R
			if ((x < 6) && (board[x + 1 + y * l_width] == opposite)) {
				numCapture += 1;
				newX = x + 2;
				while (newX <= 7) {
					nextSpace = board[newX + y * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX += 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + 1 + y * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// DoR
			if ((x < 6) && (y > 1)
					&& (board[x + 1 + (y - 1) * l_width] == opposite)) {
				numCapture += 1;
				newX = x + 2;
				newY = y - 2;
				while ((newX <= 7) && (newY >= 0)) {
					nextSpace = board[newX + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX += 1;
					newY -= 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + i + (y - i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// Do
			if ((y > 1) && (board[x + (y - 1) * l_width] == opposite)) {
				numCapture += 1;
				newY = y - 2;
				while (newY >= 0) {
					nextSpace = board[x + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newY -= 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x + (y - i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// DoL
			if ((x > 1) && (y > 1)
					&& (board[x - 1 + (y - 1) * l_width] == opposite)) {
				numCapture += 1;
				newX = x - 2;
				newY = y - 2;
				while ((newX >= 0) && (newY >= 0)) {
					nextSpace = board[newX + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX -= 1;
					newY -= 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x - i + (y - i) * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// L
			if ((x > 1) && (board[x - 1 + y * l_width] == opposite)) {
				numCapture += 1;
				newX = x - 2;
				while (newX >= 0) {
					nextSpace = board[newX + y * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX -= 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x - i + y * l_width] = colorTP;
			}
			numCapture = 0;
			captureFlag = false;

			// UpL
			if ((x > 1) && (y < 6)
					&& (board[x - 1 + (y + 1) * l_width] == opposite)) {
				numCapture += 1;
				newX = x - 2;
				newY = y + 2;
				while ((newX >= 0) && (newY <= 7)) {
					nextSpace = board[newX + newY * l_width];
					if (nextSpace == colorTP) {
						captureFlag = true;
						break;
					} else if (nextSpace == 0) {
						break;
					} else {
						numCapture += 1;
					}
					newX -= 1;
					newY += 1;
				}
			}
			if (!captureFlag) {
				numCapture = 0;
			}
			for (i = 1; i < numCapture + 1; i++) {
				board[x - i + (y + i) * l_width] = colorTP;
			}

			//check end
			end = true;
			for (i = 0; i < l_width * l_width; i++) {
				if (tempBoard[i] == 0) {
					end = false;
					break;
				}
			}

		}
		if (colorTP == 1) {
			colorTP = 2;
		} else {
			colorTP = 1;
		}
	}
	black = 0;
	white = 0;
	for (int i = 0; i < l_width * l_width; i++) {
		if (tempBoard[i] == 1) {
			black += 1;
		} else if (tempBoard[i] == 2) {
			white += 1;
		}
	}
	int win = 0;
	if (black > white) {
		win = 1;
	} else if (white > black) {
		win = 2;
	}

	if (win == *colorToPlay) {
		atomicAdd(wins, (float) 1.0);
	} else if (win == 0) {
		atomicAdd(wins, (float) 0.5);
	}
}

